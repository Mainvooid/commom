#include "hip/hip_runtime.h"
﻿#include <common/cuda/texture_reference.cuh>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <hip/channel_descriptor.h>

#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/hip_texture_types.h>

namespace common {
    namespace cuda {

        //纹理参照系必须定义在所有函数体外,需要显式声明,用NVCC编译,不支持3元组
        texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef_2d_uchar4;

        template<class T, int texType, enum hipTextureReadMode mode >
        hipError_t cuda_get_texture_reference(const textureReference ** texref)
        {
            //TODO 需要根据texType进行对象选择(texture需要显式定义,若为模板,在release模式下会初始化失败),暂只支持2d_uchar4
            return hipGetTextureReference(texref, HIP_SYMBOL(&texRef_2d_uchar4));
        }

        hipError_t cuda_get_texture_reference_2d_uchar4(const textureReference ** texref)
        {
            return cuda_get_texture_reference<uchar4, hipTextureType2D, hipReadModeElementType>(texref);
        }

    }// namespace cuda
} // namespace common