#include "hip/hip_runtime.h"
﻿#include <common/cuda/texture_reference.hpp>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <hip/channel_descriptor.h>
#include <type_traits>

#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/hip_texture_types.h>

namespace common {
    namespace cuda {

        //纹理参照系必须定义在所有函数体外(全局性),需要显式声明,用NVCC编译,不支持3元组
        texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef_2d_uchar4;
        texture<float4, hipTextureType2D, hipReadModeElementType> texRef_2d_float4;

        hipError_t cuda_get_texture_reference_2d_uchar4(const textureReference ** texref) {
            return hipGetTextureReference(texref, HIP_SYMBOL(&texRef_2d_uchar4));
        }
        hipError_t cuda_get_texture_reference_2d_float4(const textureReference ** texref) {
            return hipGetTextureReference(texref, HIP_SYMBOL(&texRef_2d_float4));
        }
    }// namespace cuda
} // namespace common